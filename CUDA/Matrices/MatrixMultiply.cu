
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;
using namespace chrono;


#define CUDA_CHECK_ERROR(call) {  \
    hipError_t ERROR = call; \
    if(ERROR != hipSuccess) { \
        cerr << "CUDA ERROR: " << hipGetErrorString(ERROR) << " in file " << __FILE__ << " at line " << __LINE__ << endl; \
        exit(EXIT_FAILURE); \
    } \
    else { \
        cout << "CUDA CALL SUCCESSFULL: " << #call << " in file " << __FILE__ << " at line " << __LINE__ << endl; \
    } \
}  \

// naive version of matrix multiplication on CUDA, we will implement optimized version soon
__global__ void MatrixMult(int* A, int* B, int* C, int N, int K, int M) {
    int curr_row = blockIdx.y * blockDim.y + threadIdx.y;
    int curr_col = blockIdx.x * blockDim.x + threadIdx.x;

    int sum = 0;
    if(curr_row < N and curr_col < M) {
        for(int i = 0;i < K; i++) {
            sum += A[curr_row * K + i] * B[i * M + curr_col];
        }
        C[curr_row * M + curr_col] = sum;
    }
}

void MatrixMultOnCPU(int* A, int* B, int* C, int N, int K, int M) {
    for(int i = 0;i < N; i++) {
        for(int j = 0; j < M; j++) {
            int sum = 0;
            
            for(int k = 0;k < K; k++) {
                sum += A[i*K + k] * B[k*M + j];
            }
            C[i*M + j] = sum;

        }
    }

    return;
}

void fill(int* M, int N) {
    for(int i = 0;i < N; i++) M[i] = rand() % 20;
    return;
}

void display(int* A, int rows, int cols) {
    for(int i = 0;i < rows; i++) {
        for(int j = 0;j < cols; j++) {
            cout << A[i*cols + j] << " ";
        }
        cout << endl;
    }
}

high_resolution_clock::time_point getTime() {
    return high_resolution_clock::now();
}

int main() {
    int N = 1000;
    int K = 1000;
    int M = 1000;

    size_t sizeA = N*K*sizeof(int);
    size_t sizeB = K*M*sizeof(int);
    size_t sizeC = N*M*sizeof(int);

    int *A = (int*)malloc(sizeA), *B = (int*)malloc(sizeB), *C = (int*)malloc(sizeC);

    fill(A,N*K);
    fill(B,K*M);

    int *dA, *dB, *dC;
    CUDA_CHECK_ERROR(hipMalloc((void**)&dA,sizeA)); CUDA_CHECK_ERROR(hipMalloc((void**)&dB,sizeB)); CUDA_CHECK_ERROR(hipMalloc((void**)&dC,sizeC));
    CUDA_CHECK_ERROR(hipMemcpy(dA,A,sizeA,hipMemcpyHostToDevice)); CUDA_CHECK_ERROR(hipMemcpy(dB,B,sizeB,hipMemcpyHostToDevice));

    // Invoking the kernel
    dim3 threadsPerBlock(2,2,1);

    int blocksInX = (M + threadsPerBlock.x - 1) / threadsPerBlock.x;
    int blocksInY = (N + threadsPerBlock.y - 1) / threadsPerBlock.y;
    dim3 gridSize(blocksInX,blocksInY,1);

    auto s = getTime(); 
    MatrixMult<<<gridSize,threadsPerBlock>>> (dA,dB,dC,N,K,M); CUDA_CHECK_ERROR(hipDeviceSynchronize());
    auto e = getTime();
    

    milliseconds durationGPU = duration_cast<milliseconds> (e - s);
    cout << "Time taken for CUDA Kernel: " << durationGPU.count() << " milliseconds" << endl;

    CUDA_CHECK_ERROR(hipMemcpy(C,dC,sizeC,hipMemcpyDeviceToHost));
    CUDA_CHECK_ERROR(hipFree(dA)); CUDA_CHECK_ERROR(hipFree(dB)); CUDA_CHECK_ERROR(hipFree(dC));

    // int* C_CPU = (int*)malloc(sizeC);

    // s = getTime();
    // MatrixMultOnCPU(A,B,C_CPU,N,K,M);
    // e = getTime();

    // nanoseconds durationCPU = duration_cast<nanoseconds> (e - s);
    // cout << "Time taken for CPU: " << durationCPU.count() << endl;

    // cout << "Matrix A: " << endl; display(A,N,K);
    // cout << "Matrix B: " << endl; display(B,K,M);
    
    // cout << "Matrix C from GPU: " << endl; display(C,N,M);
    // cout << "Matrix C from CPU: " << endl; display(C_CPU,N,M);

    // cout << "SpeedUP: " << (float) (durationCPU.count()) / (durationGPU.count() * 1.0) << endl; // Around 47 times faster

    free(A); free(B); free(C);
    return 0;
}